#include "hip/hip_runtime.h"
#include "thesis/batched_fft.h"

#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

#ifdef USING_CUDA

__global__ void multiply(int N, int batch, hipDoubleComplex *inp, int *id,
                         hipDoubleComplex *out) {
  int _N = blockIdx.x * blockDim.x + threadIdx.x;
  int _batch = blockIdx.y * blockDim.y + threadIdx.y;
  if (_batch < batch && _N < N) {
    int left = id[_batch * 2];
    int right = id[_batch * 2 + 1];
    out[_batch * N + _N] = hipCmul(inp[left * N + _N], inp[right * N + _N]);
  }
}

namespace thesis {

class CuFFT : public BatchedFFT {
private:
  int _isInitCode;
  int *_multiplication_pair_ptr;
  hipfftDoubleComplex *_inp_data;
  hipfftDoubleComplex *_out_data;
  hipfftHandle _inp_plan;
  hipfftHandle _out_plan;

public:
  // Constructors
  CuFFT() = delete;
  CuFFT(const CuFFT &) = delete;
  CuFFT(int N, int batch, int cache) : BatchedFFT(N, batch, cache) {
    _isInitCode = 0;
    _multiplication_pair_ptr = nullptr;
    _inp_data = nullptr;
    _out_data = nullptr;
    _inp_plan = 0;
    _out_plan = 0;
  }

  void clean() {
    if (_isInitCode & 16) {
      hipfftDestroy(_out_plan);
      _out_plan = 0;
    }
    if (_isInitCode & 8) {
      hipfftDestroy(_inp_plan);
      _inp_plan = 0;
    }
    if (_isInitCode & 4) {
      hipFree(_out_data);
      _out_data = nullptr;
    }
    if (_isInitCode & 2) {
      hipFree(_inp_data);
      _inp_data = nullptr;
    }
    if (_isInitCode & 1) {
      hipFree(_multiplication_pair_ptr);
      _multiplication_pair_ptr = nullptr;
    }
    _isInitCode = 0;
  }
  bool init() {
#if defined(USING_32BIT)
    const int mode = 4;
#else
    const int mode = 8;
#endif
    if (hipMalloc(&_multiplication_pair_ptr, sizeof(int) * _batch * 2) ==
        hipSuccess)
      _isInitCode |= 1;
    if (hipMalloc(&_inp_data, sizeof(hipfftDoubleComplex) * (_N * mode + 1) *
                                   (_batch + _cache)) == hipSuccess)
      _isInitCode |= 2;
    if (hipMalloc(&_out_data, sizeof(hipfftDoubleComplex) * (_N * mode + 1) *
                                   _batch) == hipSuccess)
      _isInitCode |= 4;
    if (hipfftPlan1d(&_inp_plan, _N * 2 * mode, HIPFFT_D2Z, _batch) !=
        HIPFFT_SUCCESS)
      _isInitCode |= 8;
    if (hipfftPlan1d(&_out_plan, _N * 2 * mode, HIPFFT_Z2D, _batch) !=
        HIPFFT_SUCCESS)
      _isInitCode |= 16;
    if (_isInitCode == 31)
      return true;
    clean();
    return false;
  }

  // Destructor
  ~CuFFT() { clean(); }

  // Copy assignment operator
  using BatchedFFT::operator=;
  CuFFT &operator=(const CuFFT &obj) = delete;

  using BatchedFFT::doFFT;
  bool doFFT() {
#if defined(USING_32BIT)
    const int mode = 4;
#else
    const int mode = 8;
#endif
    if (_isInitCode == 0)
      return false;
    if (hipMemcpy(_inp_data, _inp.data(),
                   sizeof(hipfftDoubleReal) * _N * 2 * mode * _batch,
                   hipMemcpyHostToDevice) != hipSuccess)
      return false;
    if (hipfftExecD2Z(_inp_plan, (hipfftDoubleReal *)_inp_data, _inp_data) !=
        HIPFFT_SUCCESS)
      return false;
    if (hipDeviceSynchronize() != hipSuccess)
      return false;
    if (hipMemcpy(_fft_inp.data(), _inp_data,
                   sizeof(hipfftDoubleComplex) * (_N * mode + 1) * _batch,
                   hipMemcpyDeviceToHost) != hipSuccess)
      return false;
    return true;
  }
  using BatchedFFT::doMultiplicationAndIFFT;
  bool doMultiplicationAndIFFT() {
#if defined(USING_32BIT)
    const int mode = 4;
#else
    const int mode = 8;
#endif
    if (_isInitCode == 0)
      return false;
    if (hipMemcpy(_multiplication_pair_ptr, _multiplication_pair.data(),
                   sizeof(int) * _batch * 2,
                   hipMemcpyHostToDevice) != hipSuccess)
      return false;
    if (hipMemcpy(_inp_data, _fft_inp.data(),
                   sizeof(hipfftDoubleComplex) * (_N * mode + 1) *
                       (_batch + _cache),
                   hipMemcpyHostToDevice) != hipSuccess)
      return false;
    int threadsPerBlock = 512;
    // _N * mode + 512 = (_N * mode + 1) + (512 - 1)
    dim3 numBlocks((_N * mode + 512) / 512, _batch);
    multiply<<<numBlocks, threadsPerBlock>>>(
        _N * mode + 1, _batch, _inp_data, _multiplication_pair_ptr, _out_data);
    if (hipfftExecZ2D(_out_plan, _out_data, (hipfftDoubleReal *)_out_data) !=
        HIPFFT_SUCCESS)
      return false;
    if (hipDeviceSynchronize() != hipSuccess)
      return false;
    if (hipMemcpy(_out.data(), _out_data,
                   sizeof(hipfftDoubleReal) * _N * 2 * mode * _batch,
                   hipMemcpyDeviceToHost) != hipSuccess)
      return false;
    return true;
  }
};

BatchedFFT *BatchedFFT::newCustomInstance(int N, int batch, int cache) {
  CuFFT *obj = new CuFFT(N, batch, cache);
  if (obj->init())
    return obj;
  delete obj;
  return nullptr;
}

} // namespace thesis

#endif
