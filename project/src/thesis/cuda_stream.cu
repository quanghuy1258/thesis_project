
#include <hip/hip_runtime.h>
#ifdef USING_CUDA

#include "thesis/stream.h"

namespace thesis {

void *Stream::cudaCreateS() {
  cudaStream_t *streamPtr = (cudaStream_t *)malloc(sizeof(cudaStream_t));
  if (cudaStreamCreate(streamPtr) != cudaSuccess) {
    free(streamPtr);
    return nullptr;
  }
  return streamPtr;
}
void Stream::cudaSynchronizeS(void *streamPtr) {
  if (streamPtr) {
    cudaStream_t *s = (cudaStream_t *)streamPtr;
    cudaStreamSynchronize(*s);
  }
}
void Stream::cudaDestroyS(void *streamPtr) {
  if (streamPtr) {
    cudaStream_t *s = (cudaStream_t *)streamPtr;
    cudaStreamDestroy(*s);
    free(s);
  }
}

} // namespace thesis

#endif
