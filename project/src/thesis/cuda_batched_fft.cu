
#include <hip/hip_runtime.h>
#ifdef USING_CUDA

#include "thesis/batched_fft.h"

#include <cuComplex.h>
#include <cufft.h>

namespace thesis {

#if defined(USING_32BIT)
const int mode = 4;
#else
const int mode = 8;
#endif

__global__ void _expand(int N, TorusInteger *pol, void *data_inp) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int m = blockIdx.y * blockDim.y + threadIdx.y;
  if (n < N && m < mode) {
    double *double_ptr = (double *)data_inp;
    if (m < mode / 2) {
      TorusInteger num = pol[n];
      for (int i = 0; i < m; i++)
        num >>= 16;
      double_ptr[n * mode + m] = num & 0xFFFF;
      double_ptr[n * mode + m] /= 2.0;
      double_ptr[(n + N) * mode + m] = -double_ptr[n * mode + m];
    } else {
      double_ptr[n * mode + m] = 0.0;
      double_ptr[(n + N) * mode + m] = -double_ptr[n * mode + m];
    }
  }
}

__global__ void _multiply(int length, void *left, void *right, void *result) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < length) {
    cuDoubleComplex *_left = (cuDoubleComplex *)left;
    cuDoubleComplex *_right = (cuDoubleComplex *)right;
    cuDoubleComplex *_result = (cuDoubleComplex *)result;
    _result[i] = cuCmul(_left[i], _right[i]);
  }
}

__global__ void _collapse(int N, void *data_mul) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    TorusInteger *torus_ptr = (TorusInteger *)data_mul;
    double *double_ptr = (double *)data_mul;
    TorusInteger num = 0;
    for (int i = mode / 2 - 1; i >= 0; i--) {
      num <<= 16;
      num += llround(-double_ptr[(n + N) * mode + i] / (N * mode));
    }
    torus_ptr[n] = num;
  }
}

__global__ void _add(int N, int col, TorusInteger *pol, void *data_mul) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    cuDoubleComplex *_data_mul = (cuDoubleComplex *)data_mul;
    for (int i = 0; i < col; i++) {
      TorusInteger *torus_ptr =
          (TorusInteger *)(_data_mul + (N * mode + 1) * i);
      pol[n] += torus_ptr[n];
    }
  }
}

__global__ void _sub(int N, int col, TorusInteger *pol, void *data_mul) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    cuDoubleComplex *_data_mul = (cuDoubleComplex *)data_mul;
    for (int i = 0; i < col; i++) {
      TorusInteger *torus_ptr =
          (TorusInteger *)(_data_mul + (N * mode + 1) * i);
      pol[n] -= torus_ptr[n];
    }
  }
}

void BatchedFFT::cudaCreatePlan() {
  _plan_inp.resize((_row + 1) * _col);
  for (int i = 0; i < (_row + 1) * _col; i++) {
    cufftHandle *cufftHandle_ptr =
        (cufftHandle *)std::malloc(sizeof(cufftHandle));
    cudaStream_t *cudaStream_t_ptr = (cudaStream_t *)_stream_inp[i];
    if (cufftPlan1d(cufftHandle_ptr, _N * 2 * mode, CUFFT_D2Z, 1) !=
        CUFFT_SUCCESS) {
      std::free(cufftHandle_ptr);
      _plan_inp[i] = nullptr;
      throw std::runtime_error("Cannot create cufftPlan1d");
    } else {
      _plan_inp[i] = cufftHandle_ptr;
      cufftSetStream(*cufftHandle_ptr, *cudaStream_t_ptr);
    }
  }
  _plan_mul.resize(_row * _col);
  for (int i = 0; i < _row * _col; i++) {
    cufftHandle *cufftHandle_ptr =
        (cufftHandle *)std::malloc(sizeof(cufftHandle));
    cudaStream_t *cudaStream_t_ptr = (cudaStream_t *)_stream_mul[i];
    if (cufftPlan1d(cufftHandle_ptr, _N * 2 * mode, CUFFT_Z2D, 1) !=
        CUFFT_SUCCESS) {
      std::free(cufftHandle_ptr);
      _plan_mul[i] = nullptr;
      throw std::runtime_error("Cannot create cufftPlan1d");
    } else {
      _plan_mul[i] = cufftHandle_ptr;
      cufftSetStream(*cufftHandle_ptr, *cudaStream_t_ptr);
    }
  }
}
void BatchedFFT::cudaDestroyPlan() {
  for (int i = 0; i < (_row + 1) * _col; i++) {
    if (_plan_inp[i]) {
      cufftHandle *cufftHandle_ptr = (cufftHandle *)_plan_inp[i];
      cufftDestroy(*cufftHandle_ptr);
      std::free(cufftHandle_ptr);
    }
  }
  for (int i = 0; i < _row * _col; i++) {
    if (_plan_mul[i]) {
      cufftHandle *cufftHandle_ptr = (cufftHandle *)_plan_mul[i];
      cufftDestroy(*cufftHandle_ptr);
      std::free(cufftHandle_ptr);
    }
  }
}
void BatchedFFT::cudaSetInp(TorusInteger *pol, int r, int c) {
  int threadsPerBlock = 512;
  // _N * mode + 512 = (_N * mode + 1) + (512 - 1)
  dim3 numBlocks((_N * mode + 512) / 512, mode);
  cudaStream_t *cudaStream_t_ptr = (cudaStream_t *)_stream_inp[r * _col + c];
  _expand<<<numBlocks, threadsPerBlock, 0, *cudaStream_t_ptr>>>(
      _N, pol, _data_inp[r * _col + c]);
  cufftExecD2Z(*(cufftHandle *)_plan_inp[r * _col + c],
               (double *)_data_inp[r * _col + c],
               (cuDoubleComplex *)_data_inp[r * _col + c]);
}
void BatchedFFT::cudaSetInp(TorusInteger *pol, int c) {
  int threadsPerBlock = 512;
  // _N * mode + 512 = (_N * mode + 1) + (512 - 1)
  dim3 numBlocks((_N * mode + 512) / 512, mode);
  cudaStream_t *cudaStream_t_ptr = (cudaStream_t *)_stream_inp[_row * _col + c];
  _expand<<<numBlocks, threadsPerBlock, 0, *cudaStream_t_ptr>>>(
      _N, pol, _data_inp[_row * _col + c]);
  cufftExecD2Z(*(cufftHandle *)_plan_inp[_row * _col + c],
               (double *)_data_inp[_row * _col + c],
               (cuDoubleComplex *)_data_inp[_row * _col + c]);
}
void BatchedFFT::cudaSetMul(int r, int c) {
  int threadsPerBlock = 512;
  // _N * mode + 512 = (_N * mode + 1) + (512 - 1)
  int numBlocks = (_N * mode + 512) / 512;
  cudaStream_t *cudaStream_t_ptr = (cudaStream_t *)_stream_mul[r * _col + c];
  _multiply<<<numBlocks, threadsPerBlock, 0, *cudaStream_t_ptr>>>(
      _N * mode + 1, _data_inp[r * _col + c], _data_inp[_row * _col + c],
      _data_mul[r * _col + c]);
  cufftExecZ2D(*(cufftHandle *)_plan_mul[r * _col + c],
               (cuDoubleComplex *)_data_mul[r * _col + c],
               (double *)_data_mul[r * _col + c]);
  _collapse<<<numBlocks, threadsPerBlock, 0, *cudaStream_t_ptr>>>(
      _N, _data_mul[r * _col + c]);
}
void BatchedFFT::cudaAddAllOut(TorusInteger *pol, int r) {
  int threadsPerBlock = 512;
  // _N + 511 = _N + (512 - 1)
  int numBlocks = (_N + 511) / 512;
  cudaStream_t *cudaStream_t_ptr = (cudaStream_t *)_stream_out[r];
  _add<<<numBlocks, threadsPerBlock, 0, *cudaStream_t_ptr>>>(
      _N, _col, pol, _data_mul[r * _col]);
}
void BatchedFFT::cudaSubAllOut(TorusInteger *pol, int r) {
  int threadsPerBlock = 512;
  // _N + 511 = _N + (512 - 1)
  int numBlocks = (_N + 511) / 512;
  cudaStream_t *cudaStream_t_ptr = (cudaStream_t *)_stream_out[r];
  _sub<<<numBlocks, threadsPerBlock, 0, *cudaStream_t_ptr>>>(
      _N, _col, pol, _data_mul[r * _col]);
}

} // namespace thesis

#endif
